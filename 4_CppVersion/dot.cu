#include <iostream>
#include <iomanip>
#include <new>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <limits.h>

#define imin(a,b) (a<b?a:b)

const int N = 1<<15;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

// 计时器函数
double cpuSeconds()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

// 使用交错配对进行点积运算
__global__ void dotOnGPU1(float *devA, float *devB, float *devC)
{
    __shared__ float cache[threadsPerBlock]; // 保存每个 thread 计算的加和值
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // 数组索引
    int cacheIndex = threadIdx.x; // 每个 block 中的 thread 索引

    float temp = 0;
    while(tid < N) {
        temp += devA[tid] * devB[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while(i != 0)
    {
        if(cacheIndex < i)
        {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if(cacheIndex == 0)
    {
        devC[blockIdx.x] = cache[0];
    }
}

int main(int argc, char const *argv[])
{
    float *hostA, *hostB, *partialC;
    float hostC;
    float *devA, *devB, *devPartialC;

    hostA = new float[N];
    hostB = new float[N];
    partialC = new float[blocksPerGrid];

    size_t dataSize = N*sizeof(float);

    hipMalloc((void**)&devA, dataSize);
    hipMalloc((void**)&devB, dataSize);
    hipMalloc((void**)&devPartialC, blocksPerGrid*sizeof(float));

    for (int i = 0; i < N; ++i)
    {
        hostA[i] = i;
        hostB[i] = i * 2;
    }

    hipMemcpy(devA, hostA, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(devB, hostB, dataSize, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double tStart = cpuSeconds();
    dotOnGPU1<<<blocksPerGrid, threadsPerBlock>>>(devA, devB, devPartialC);
    hipDeviceSynchronize();
    double tElaps = cpuSeconds() - tStart;
    std::cout.precision(20);
    std::cout << "Time: " << std::setw(25) << tElaps*1000.0f << "ms.  ";

    hipMemcpy(partialC, devPartialC, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

    hostC = 0;
    for (int i = 0; i < blocksPerGrid; ++i)
    {
        hostC += partialC[i];
    }

    std::cout << "GPU Result: " << std::setw(25) << hostC << std::endl;

    hipFree(devA);
    hipFree(devB);
    hipFree(devPartialC);

    tStart = cpuSeconds();
    float sum = 0;
    for (int i = 0; i < N; ++i)
    {
        sum += hostA[i] * hostB[i];
    }
    tElaps = cpuSeconds() - tStart;
    std::cout << "Time: " << std::setw(25) << tElaps*1000.0f << "ms.  ";
    std::cout << "CPU Result: " << std::setw(25) << sum << std::endl;

    free(hostA);
    free(hostB);
    free(partialC);

    return 0;
}