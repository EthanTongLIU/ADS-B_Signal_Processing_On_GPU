#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "myProject.h"

const int N = 33 * 1024;
const int threadsPerBlock = 256;

// dot between 2 vectors
__global__ void dot(float *a, float *b, float *c)
{
    __shared__ float cache[threadsPerBlock]; // 名为 cache 的共享内存缓冲区，用于保存每个线程计算的加和值，数组大小声明为 threadsPerBlock
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while(tid < N)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // 设置 cache 中相应位置上的值
    cache[cacheIndex] = temp;
}