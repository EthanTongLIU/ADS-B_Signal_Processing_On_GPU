#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

// 打印数组
void display_array(float *array, int N)
{
    std::cout << "[ ";
    for (int i = 0; i < N; ++i)
    {
        std::cout << array[i] << " ";
    }
    std::cout << "]" << std::endl;
}

// 初始化数据
void initial_data(float *ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));

    for(int i = 0; i < size; ++i)
    {
        ip[i] = (float)(rand()&0xFF)/10.0f;
    }
}

// GPU 点积运算
__global__ void dot_on_gpu(float *dev_a, float *dev_b, float *dev_c, float *global_odata, unsigned int N)
{
    // 向量加法
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dev_c[idx] = dev_a[idx] + dev_b[idx];
    __syncthreads();

    // 归约运算
    unsigned int tid = threadIdx.x;

    // 找到指向每个 block 的指针
    float *idata = dev_c + blockIdx.x * blockDim.x;

    // 边界检查
    if (idx >= N) return;

    // 局部归约
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

// GPU 数组加法
__global__ void sum_arrays_on_gpu(float *dev_a, float *dev_b, float *dev_c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    dev_c[i] = dev_a[i] + dev_b[i];
}

// 在 CPU 上递归归约
float recursive_reduce(float *array, int N)
{
    float sum = 0;
    for (int i = 0; i < N; ++i)
    {
        sum += array[i];
    }
    return sum;
}

// 在 GPU 上进行相邻归约（有线程束分化）
__global__ void reduce_neighbored(float *global_idata, float *global_odata, unsigned int N)
{
    // 设置线程
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 找到指向每个 block 的指针
    float *idata = global_idata + blockIdx.x * blockDim.x;

    // 边界检查
    if (idx >= N) return;

    // 局部归约
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

int main()
{
    std::cout << "Strating...\n";

    // 设置设备
    int dev = 0;
    hipSetDevice(dev);

    // 设置数组大小
    int N = 1<<24;

    // 指定 GPU 维度
    dim3 block(512, 1);
    dim3 grid((N+block.x-1)/block.x, 1);

    // 分配 host 内存
    size_t data_size = N*sizeof(float);

    float *host_a, *host_b, *h_odata;
    host_a = (float*)malloc(data_size);
    host_b = (float*)malloc(data_size);
    h_odata = (float *)malloc(grid.x * sizeof(float));

    // 给 host 内存赋值
    initial_data(host_a, N);
    initial_data(host_b, N);

    memset(h_odata, 0, grid.x);

    // 分配 device global 内存
    float *dev_a, *dev_b, *dev_c, *global_odata;
    hipMalloc((float**)&dev_a, data_size);
    hipMalloc((float**)&dev_b, data_size);
    hipMalloc((float**)&dev_c, data_size);
    hipMalloc((float**)&global_odata, grid.x * sizeof(float));

    // 从 host 向 device 复制数据
    hipMemcpy(dev_a, host_a, data_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, data_size, hipMemcpyHostToDevice);

    // 在 host 端调用核函数
    // sum_arrays_on_gpu<<<grid, block>>>(dev_a, dev_b, dev_c);
    dot_on_gpu<<<grid, block>>>(dev_a, dev_b, dev_c, global_odata, N);

    // 从 device 向 host 复制数据
    // cudaMemcpy(gpu_ref, dev_c, data_size, cudaMemcpyDeviceToHost);
    hipMemcpy(h_odata, global_odata, grid.x * sizeof(float), hipMemcpyDeviceToHost);

    // 显示运算结果
    // display_array(gpu_ref, N);

    // 在 CPU 上进行最后的归约
    float gpu_result = recursive_reduce(h_odata, grid.x);
    std::cout << "GPU dot result: " << gpu_result << std::endl;

    // 在 CPU 上执行全部归约
    float cpu_result = 0;
    for (int i = 0; i < N; ++i)
    {
        cpu_result += host_a[i] * host_b[i];
    }
    std::cout << "CPU dot result: " << cpu_result << std::endl;

    // 释放 device 内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(global_odata);

    // 释放 host 内存
    free(host_a);
    free(host_b);
    free(h_odata);

    std::cout << "End...\n";

    return 0;
}