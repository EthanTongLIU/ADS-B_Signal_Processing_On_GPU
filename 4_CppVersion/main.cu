#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

// 计时器函数
double cpu_seconds()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

// 打印数组
void display_array(unsigned int *array, unsigned int N)
{
    std::cout << "[ ";
    for (int i = 0; i < N; i++)
    {
        std::cout << array[i] << " ";
    }
    std::cout << "]" << std::endl;
}

// 初始化数据
void initial_data(unsigned int *ip, unsigned int size)
{
    // time_t t;
    // srand((unsigned) time(&t));

    for(int i = 0; i < size; i++)
    {
        ip[i] = (unsigned int)( (rand() & 0xFF) / 30 );
    }
}

// GPU 数组加法
__global__ void sum_arrays_on_gpu(unsigned int *dev_a, unsigned int *dev_b, unsigned int *dev_c)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    dev_c[i] = dev_a[i] + dev_b[i];
}

// GPU 数组乘法
__global__ void multi_arrays_on_gpu(unsigned int *dev_a, unsigned int *dev_b, unsigned int *dev_c)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    dev_c[i] = dev_a[i] * dev_b[i];
}

// 在 CPU 上递归归约
int recursive_reduce(unsigned int *array, unsigned int N)
{
    unsigned int sum = 0;
    for (int i = 0; i < N; i++)
    {
        sum += array[i];
    }
    return sum;
}

// 在 GPU 上进行相邻归约（有线程束分化）
__global__ void reduce_neighbored(unsigned int *global_idata, unsigned int *global_odata, unsigned int N)
{
    // 设置线程
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 找到指向每个 block 的指针
    unsigned int *idata = global_idata + blockIdx.x * blockDim.x;

    // 边界检查
    if (idx >= N) return;

    // block 内归约
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

// GPU 点积运算（有 wrap 分化的相邻归约）
__global__ void dot_on_gpu_1(unsigned int *dev_a, unsigned int *dev_b, unsigned int *dev_c, unsigned int *global_odata, unsigned int N)
{
    // 线程 id
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 向量乘法
    dev_c[idx] = dev_a[idx] * dev_b[idx];
    __syncthreads();

    // 每个 block 中的 thread ID
    unsigned int tid = threadIdx.x;

    // 找到指向每个 block 的指针
    unsigned int *idata = dev_c + blockIdx.x * blockDim.x;

    // 边界检查
    if (idx >= N) return;

    // block 内归约
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

// GPU 点积运算（减少 wrap 分化的相邻归约）
__global__ void dot_on_gpu_2(unsigned int *dev_a, unsigned int *dev_b, unsigned int *dev_c, unsigned int *global_odata, unsigned int N)
{
    // 线程 id
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 向量乘法
    dev_c[idx] = dev_a[idx] * dev_b[idx];
    __syncthreads();

    // 每个 block 中的 thread ID
    unsigned int tid = threadIdx.x;

    // 找到指向每个 block 的指针
    unsigned int *idata = dev_c + blockIdx.x * blockDim.x;

    // 边界检查
    if (idx >= N) return;

    // block 内归约
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        // 将连续的 tid 映射到需要配对的元素上
        unsigned int index = 2 * stride * tid;
        if (index < blockDim.x)
        {
            idata[index] += idata[index + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

// GPU 点积运算（交错配对的归约）
__global__ void dot_on_gpu_3(unsigned int *dev_a, unsigned int *dev_b, unsigned int *dev_c, unsigned int *global_odata, unsigned int N)
{
    // 线程 id
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 向量乘法
    dev_c[idx] = dev_a[idx] * dev_b[idx];
    __syncthreads();

    // 每个 block 中的 thread ID
    unsigned int tid = threadIdx.x;

    // 找到指向每个 block 的指针
    unsigned int *idata = dev_c + blockIdx.x * blockDim.x;

    // 边界检查
    if (idx >= N) return;

    // block 内归约
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

// GPU 点积运算（展开循环 2 个数据块）
__global__ void dot_on_gpu_4(unsigned int *dev_a, unsigned int *dev_b, unsigned int *dev_c, unsigned int *global_odata, unsigned int N)
{
    // // 数组索引
    // unsigned int idx0 = blockIdx.x * blockDim.x + threadIdx.x;

    // // 向量乘法
    // dev_c[idx0] = dev_a[idx0] * dev_b[idx0];
    // __syncthreads();

    // 重新构建数组索引
    unsigned int idx = blockIdx.x + blockDim.x * 2 + threadIdx.x;

    // 每个 block 中的 thread ID
    unsigned int tid = threadIdx.x;

    // 找到指向每个 block 的指针
    unsigned int *idata = dev_c + blockIdx.x * blockDim.x * 2;

    // 展开 2 个数据块
    if (idx + blockDim.x < N)
    {
        dev_c[idx] += dev_c[idx + blockDim.x];
    }
    __syncthreads();

    // block 内归约
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

// GPU 点积运算（展开循环 8 个数据块）
__global__ void dot_on_gpu_5(unsigned int *dev_a, unsigned int *dev_b, unsigned int *dev_c, unsigned int *global_odata, unsigned int N)
{
    // // 数组索引
    // unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // // 向量乘法
    // dev_c[idx] = dev_a[idx] * dev_b[idx];
    // __syncthreads();

    // 重新构建数组索引
    unsigned int idx = blockIdx.x + blockDim.x * 8 + threadIdx.x;

    // 每个 block 中的 thread ID
    unsigned int tid = threadIdx.x;

    // 找到指向每个 block 的指针
    unsigned int *idata = dev_c + blockIdx.x * blockDim.x * 8;

    // 展开 8 个数据块
    if (idx + 7 * blockDim.x < N)
    {
        int a1 = dev_c[idx];
        int a2 = dev_c[idx + blockDim.x];
        int a3 = dev_c[idx + 2 * blockDim.x];
        int a4 = dev_c[idx + 3 * blockDim.x];
        int b1 = dev_c[idx + 4 * blockDim.x];
        int b2 = dev_c[idx + 5 * blockDim.x];
        int b3 = dev_c[idx + 6 * blockDim.x];
        int b4 = dev_c[idx + 7 * blockDim.x];
        dev_c[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    __syncthreads();

    // block 内归约
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // 将每个 block 归约后的数据赋给小全局内存
    if (tid == 0) global_odata[blockIdx.x] = idata[0];
}

int main()
{
    std::cout << "Strating...\n";

    // 设置设备
    int dev = 0;
    hipSetDevice(dev);

    // 设置数组大小
    unsigned int N = 1<<24;

    // 指定 GPU 维度
    dim3 block(512, 1);
    dim3 grid((N+block.x-1)/block.x, 1);
    std::cout << "grid " << grid.x << " block " << block.x << std::endl;

    // 分配 host 内存
    size_t data_size = N*sizeof(int);

    unsigned int *host_a, *host_b, *gpu_ref, *h_odata;
    host_a = (unsigned int*)malloc(data_size);
    host_b = (unsigned int*)malloc(data_size);
    gpu_ref = (unsigned int*)malloc(data_size);
    h_odata = (unsigned int *)malloc(grid.x * sizeof(unsigned int));

    // 给 host 内存赋值
    initial_data(host_a, N);
    initial_data(host_b, N);
    memset(gpu_ref, 0, N);
    memset(h_odata, 0, grid.x);

    // 分配 device global 内存
    unsigned int *dev_a, *dev_b, *dev_c, *global_odata;
    hipMalloc((unsigned int**)&dev_a, data_size);
    hipMalloc((unsigned int**)&dev_b, data_size);
    hipMalloc((unsigned int**)&dev_c, data_size);
    hipMalloc((unsigned int**)&global_odata, grid.x * sizeof(unsigned int));

    // 从 host 向 device 复制数据
    hipMemcpy(dev_a, host_a, data_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, data_size, hipMemcpyHostToDevice);

    // 格式化输出性能参数
    std::cout << std::setw(30) << "Type"
        << std::setw(20) << "Result"
        << std::setw(20) << "Time(ms)"
        << std::setw(20) << "1 step acceleration"
        << std::setw(20) << "Total acceleration"
        << std::endl;

    // 在 CPU 上执行全部归约
    hipDeviceSynchronize();
    double i_start = cpu_seconds();
    unsigned int cpu_result = 0;
    for (int i = 0; i < N; i++)
    {
        cpu_result += ( host_a[i] * host_b[i] );
    }
    double i_elaps = cpu_seconds() - i_start;
    std::cout << std::setw(30) << "CPU reduce recursived"
        << std::setw(20) << cpu_result
        << std::setw(20) << i_elaps * 1000
        << std::endl;

    // // dot 1
    // cudaDeviceSynchronize();
    // i_start = cpu_seconds();
    // dot_on_gpu_1<<<grid, block>>>(dev_a, dev_b, dev_c, global_odata, N);
    // cudaDeviceSynchronize();
    // double i_elaps_1 = cpu_seconds() - i_start;
    // cudaMemcpy(h_odata, global_odata, grid.x * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    // unsigned int gpu_result_1 = recursive_reduce(h_odata, grid.x); // 在 CPU 上进行最后的归约
    // std::cout << std::setw(30) << "GPU reduce neighbored"
    //     << std::setw(20) << gpu_result_1
    //     << std::setw(20) << i_elaps_1 * 1000
    //     << std::setw(20) << i_elaps / i_elaps_1
    //     << std::setw(20) << i_elaps / i_elaps_1
    //     << std::endl;

    // // dot 2
    // cudaDeviceSynchronize();
    // i_start = cpu_seconds();
    // dot_on_gpu_2<<<grid, block>>>(dev_a, dev_b, dev_c, global_odata, N);
    // cudaDeviceSynchronize();
    // double i_elaps_2 = cpu_seconds() - i_start;
    // cudaMemcpy(h_odata, global_odata, grid.x * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    // unsigned int gpu_result_2 = recursive_reduce(h_odata, grid.x); // 在 CPU 上进行最后的归约
    // std::cout << std::setw(30) << "GPU reduce neighbored less"
    //     << std::setw(20) << gpu_result_2
    //     << std::setw(20) << i_elaps_2 * 1000
    //     << std::setw(20) << i_elaps_1 / i_elaps_2
    //     << std::setw(20) << i_elaps / i_elaps_2
    //     << std::endl;

    // dot 3
    // cudaDeviceSynchronize();
    // i_start = cpu_seconds();
    // dot_on_gpu_3<<<grid, block>>>(dev_a, dev_b, dev_c, global_odata, N);
    // cudaDeviceSynchronize();
    // double i_elaps_3 = cpu_seconds() - i_start;
    // cudaMemcpy(h_odata, global_odata, grid.x * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    // unsigned int gpu_result_3 = recursive_reduce(h_odata, grid.x); // 在 CPU 上进行最后的归约
    // std::cout << std::setw(30) << "GPU reduce interleaved"
    //     << std::setw(20) << gpu_result_3
    //     << std::setw(20) << i_elaps_3 * 1000
    //     << std::setw(20) << i_elaps / i_elaps_3
    //     << std::setw(20) << i_elaps / i_elaps_3
    //     << std::endl;

    // dot 4
    multi_arrays_on_gpu<<<grid, block>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    i_start = cpu_seconds();
    dot_on_gpu_4<<<grid.x/2, block>>>(dev_a, dev_b, dev_c, global_odata, N);
    hipDeviceSynchronize();
    double i_elaps_4 = cpu_seconds() - i_start;
    hipMemcpy(h_odata, global_odata, grid.x/2 * sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int gpu_result_4 = recursive_reduce(h_odata, grid.x/2); // 在 CPU 上进行最后的归约
    std::cout << std::setw(30) << "GPU unrolling 2 data blocks"
        << std::setw(20) << gpu_result_4
        << std::setw(20) << i_elaps_4 * 1000
        << std::setw(20) << i_elaps / i_elaps_4
        << std::setw(20) << i_elaps / i_elaps_4
        << std::endl;

    // dot 5
    multi_arrays_on_gpu<<<grid, block>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    i_start = cpu_seconds();
    dot_on_gpu_5<<<grid.x/8, block>>>(dev_a, dev_b, dev_a, global_odata, N);
    hipDeviceSynchronize();
    double i_elaps_5 = cpu_seconds() - i_start;
    hipMemcpy(h_odata, global_odata, grid.x/8 * sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int gpu_result_5 = recursive_reduce(h_odata, grid.x/8); // 在 CPU 上进行最后的归约
    std::cout << std::setw(30) << "GPU unrolling 8 data blocks"
        << std::setw(20) << gpu_result_5
        << std::setw(20) << i_elaps_5 * 1000
        << std::setw(20) << i_elaps / i_elaps_5
        << std::setw(20) << i_elaps / i_elaps_5
        << std::endl;

    // 释放 device 内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(global_odata);

    // 释放 host 内存
    free(host_a);
    free(host_b);
    free(gpu_ref);
    free(h_odata);

    std::cout << "End...\n";

    return 0;
}